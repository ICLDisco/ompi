#include "ompi_config.h"
#include "coll_adapt_cuda.cuh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <stdio.h>
#include <assert.h>
#include <stdarg.h>

static int coll_adapt_cuda_kernel_enabled = 0;
hipStream_t op_internal_stream; 
hipblasHandle_t cublas_handle;

static const char *_cudaGetErrorEnum(hipblasStatus_t error)
{
    switch (error)
    {
        case HIPBLAS_STATUS_SUCCESS:
            return "HIPBLAS_STATUS_SUCCESS";

        case HIPBLAS_STATUS_NOT_INITIALIZED:
            return "HIPBLAS_STATUS_NOT_INITIALIZED";

        case HIPBLAS_STATUS_ALLOC_FAILED:
            return "HIPBLAS_STATUS_ALLOC_FAILED";

        case HIPBLAS_STATUS_INVALID_VALUE:
            return "HIPBLAS_STATUS_INVALID_VALUE";

        case HIPBLAS_STATUS_ARCH_MISMATCH:
            return "HIPBLAS_STATUS_ARCH_MISMATCH";

        case HIPBLAS_STATUS_MAPPING_ERROR:
            return "HIPBLAS_STATUS_MAPPING_ERROR";

        case HIPBLAS_STATUS_EXECUTION_FAILED:
            return "HIPBLAS_STATUS_EXECUTION_FAILED";

        case HIPBLAS_STATUS_INTERNAL_ERROR:
            return "HIPBLAS_STATUS_INTERNAL_ERROR";
    }

    return "<unknown>";
}

int coll_adapt_cuda_init(void)
{
    int device;
    hipError_t cuda_err;

    cuda_err = hipGetDevice(&device);
    if( hipSuccess != cuda_err ) {
       // OPAL_OUTPUT_VERBOSE((0, opal_datatype_cuda_output, "Cannot retrieve the device being used. Drop CUDA support!\n"));
        return -1;
    }
    //hipStreamCreate(&op_internal_stream);
    hipblasStatus_t stat;
    stat = hipblasCreate(&cublas_handle); 
    if (stat != HIPBLAS_STATUS_SUCCESS) { 
        printf("CUBLAS initialization failed\n");
        return -1; 
    }
    hipStreamCreate(&op_internal_stream);
    coll_adapt_cuda_kernel_enabled = 1;
    hipDeviceSynchronize();
    printf("CUBLAS initialization done device %d\n", device);
    return 0;
}

int coll_adapt_cuda_fini(void)
{
    coll_adapt_cuda_kernel_enabled = 0;
    hipblasDestroy(cublas_handle);
    hipStreamDestroy(op_internal_stream);
    op_internal_stream = NULL;
    return 0;
}

void* coll_adapt_cuda_malloc(size_t size)
{
    hipError_t cuda_err;
    void *ptr = NULL;
    cuda_err = hipMalloc((void**)&ptr, size);
    if( hipSuccess != cuda_err ) {
       // OPAL_OUTPUT_VERBOSE((0, opal_datatype_cuda_output, "Cannot retrieve the device being used. Drop CUDA support!\n"));
        return NULL;
    } else {
        return ptr;
    }
}

int coll_adapt_cuda_is_gpu_buffer(const void *ptr)
{
    hipMemoryType memType;
    hipDeviceptr_t dbuf = (hipDeviceptr_t)ptr;
    int res;

    res = hipPointerGetAttribute(&memType, HIP_POINTER_ATTRIBUTE_MEMORY_TYPE, dbuf);
    if (res != hipSuccess) {
        /* If we cannot determine it is device pointer,
         * just assume it is not. */
      //  OPAL_OUTPUT_VERBOSE((1, opal_datatype_cuda_output, "!!!!!!! %p is not a gpu buffer. Take no-CUDA path!\n", ptr));
        return 0;
    }
    /* Anything but hipMemoryTypeDevice is not a GPU memory */
    return (memType == hipMemoryTypeDevice) ? 1 : 0;
}

int coll_adapt_cuda_op_sum_float(void *source, void *target, int count, void *op_stream)
{
    int is_sync = 0;
    float alpha_f = 1.0;

    if (op_stream == NULL) {
        hipblasSetStream(cublas_handle, op_internal_stream);
        is_sync = 1;
    } else {
        hipblasSetStream(cublas_handle, (hipStream_t)op_stream);
    }
    //stat = hipblasDaxpy(cublas_handle, count, &alpha, (const double *)source, 1, (double *)target, 1);
    hipblasStatus_t stat = hipblasSaxpy(cublas_handle, count, &alpha_f, (const float *)source, 1, (float *)target, 1);
    if (stat != HIPBLAS_STATUS_SUCCESS) { 
        printf("hipblasSaxpy error %s. src %p, targrt %p, count %d\n", _cudaGetErrorEnum(stat), source, target, count);
        return -1; 
    }
    if (is_sync) {
        hipStreamSynchronize(op_internal_stream);
    }
    return 1;
}